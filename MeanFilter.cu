#include "hip/hip_runtime.h"
%%writefile cudabasic.cu

#include <stdlib.h>
#include <stdio.h>

#include <string.h>

double time_h = 0;
double time_d = 0;

int numOfRounds = 1;

void meanFilter_h (unsigned char* raw_image, unsigned char* filtered_image, int img_width, int img_height, int window_size)
{
  int half_window = (window_size - 1) / 2;

  for (int i=0; i < img_height; i++)
  {
    for(int j=0; j < img_width; j++)
    {
      int left_limit, right_limit, top_limit, bottom_limit;

      if(j - half_window >= 0){
        left_limit = j-half_window;
      }else{
        left_limit = 0;
      }

            if(j + half_window <= img_width-1){
        right_limit = j + half_window;
      }else{
        right_limit = img_width-1;
      }

      if(i - half_window >= 0){
        top_limit = i - half_window;
      }else{
        top_limit = 0;
      }

            if(i + half_window <= img_height-1){
        bottom_limit = i + half_window;
      }else{
        bottom_limit = img_height-1;
      }

      double sum_r = 0, sum_g = 0, sum_b = 0;
      for(int k = top_limit; k <= bottom_limit; k++)
      {
        for(int m = left_limit; m <= right_limit; m++)
        {
           int index = (k * img_width + m) * 3;

                    // Accumulate the values of each color channel separately
                    sum_r += raw_image[index];
                    sum_g += raw_image[index + 1];
                    sum_b += raw_image[index + 2];
        }
      }
       int current_window_size = (bottom_limit - top_limit + 1) * (right_limit - left_limit + 1);

            // Calculate the mean value for each color channel
            filtered_image[(i * img_width + j) * 3] = sum_r / current_window_size;
            filtered_image[(i * img_width + j) * 3 + 1] = sum_g / current_window_size;
            filtered_image[(i * img_width + j) * 3 + 2] = sum_b / current_window_size;
    }
  }
}

__global__ void meanFilter_d (unsigned char* raw_image, unsigned char* filtered_image, int img_width, int img_height, int window_size)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

  int half_window = (window_size - 1) / 2;

  if (i < img_height && j < img_width)
  {
    int left_limit, right_limit, top_limit, bottom_limit;

    if(j - half_window >= 0){
      left_limit = j-half_window;
    }else{
      left_limit = 0;
    }

        if(j + half_window <= img_width-1){
      right_limit = j + half_window;
    }else{
      right_limit = img_width-1;
    }

    if(i - half_window >= 0){
      top_limit = i - half_window;
    }else{
      top_limit = 0;
    }

        if(i + half_window <= img_height-1){
      bottom_limit = i + half_window;
    }else{
      bottom_limit = img_height-1;
    }

    double sumR = 0, sumG = 0, sumB = 0;
    int current_window_size = 0;

    for(int k = top_limit; k <= bottom_limit; k++)
    {
      for(int m = left_limit; m <= right_limit; m++)
      {
        // Calculate the indices for Red, Green, and Blue components
        int index = 3 * (k * img_width + m);
        sumR += raw_image[index];       // Red component
        sumG += raw_image[index + 1];   // Green component
        sumB += raw_image[index + 2];   // Blue component
        current_window_size++;
      }
    }
    // Calculate the average values for Red, Green, and Blue components
        unsigned char avgR = sumR / current_window_size;
        unsigned char avgG = sumG / current_window_size;
        unsigned char avgB = sumB / current_window_size;

        // Write the average values to the filtered image
        int output_index = 3 * (i * img_width + j);
        filtered_image[output_index] = avgR;       // Red component
        filtered_image[output_index + 1] = avgG;   // Green component
        filtered_image[output_index + 2] = avgB;   // Blue component
  }
}

void saveBitmap(const char* filename, int width, int height, unsigned char* imageData,unsigned char *info) {
    FILE* file = fopen(filename, "wb");
    if (file == NULL) {
        fprintf(stderr, "Error: Unable to open file %s for writing.\n", filename);
        exit(1);
    }

    printf("File %s opened successfully for writing.\n", filename);

    // Calculate padding bytes
    int padding = (4 - (width * 3) % 4) % 4;

    unsigned char fileHeader[138];
    memcpy(fileHeader, info, 138);

  
    // Write the file header
    fwrite(fileHeader, sizeof(unsigned char), 138, file);

    // Allocate memory for a single row including padding
    unsigned char* paddedRow = (unsigned char*)malloc(sizeof(unsigned char) * (width * 3 + padding));
    if (paddedRow == NULL) {
        fprintf(stderr, "Error: Unable to allocate memory for padded row.\n");
        fclose(file);
        exit(1);
    }

    fwrite(imageData, sizeof(unsigned char), width * height * 3, file);

    // Write padding bytes
    //unsigned char paddingData[4] = {0, 0, 0, 0};
    //for (int i = 0; i < padding; i++) {
    //    fwrite(paddingData, sizeof(unsigned char), 1, file);
   // }

    if (ferror(file)) {
          fprintf(stderr, "Error writing to file.\n");
          fclose(file);
          free(paddedRow);
          exit(1);
      }

    // Free allocated memory

    // Close the file
    fclose(file);
}


int main(int argc,char **argv)
{
    printf("Begin......\n");

  //get bitmap to a char array
    FILE* file = fopen("/content/drive/MyDrive/img_640.bmp", "rb");
    unsigned char info[138];
    fread(info, sizeof(unsigned char), 138, file);

    int width, height;
    memcpy(&width, info + 18, sizeof(int));
    memcpy(&height, info + 22, sizeof(int));

    int window_size = 3;

    int size = 3 * width * abs(height);
    unsigned char *inputImage = (unsigned char*)malloc(size * sizeof(unsigned char));
    unsigned char* result_image_data_d;
    unsigned char *result_image_data_h=(unsigned char*)malloc(size * sizeof(unsigned char));
    unsigned char *result_image_data_h1=(unsigned char*)malloc(size * sizeof(unsigned char));

    unsigned char* image_data_d;

    fread(inputImage, sizeof(unsigned char), size, file);
    fclose(file);

    int block_size = 32;
    int grid_size = width/block_size;

    dim3 dimBlock(block_size, block_size, 1);
    dim3 dimGrid(grid_size, grid_size, 1);

    for(int x = 0; x < numOfRounds; x += 1)
    {
        hipMalloc((void **)&image_data_d,size*sizeof(unsigned char));
        hipMalloc((void **)&result_image_data_d,size*sizeof(unsigned char));

        hipMemcpy(image_data_d,inputImage,size*sizeof(unsigned char),hipMemcpyHostToDevice);

        clock_t start_d=clock();
    //execution of GPU code
        meanFilter_d <<< dimGrid, dimBlock >>> (image_data_d, result_image_data_d, width, height, window_size);
        hipDeviceSynchronize();


        hipError_t error = hipGetLastError();
        if(error!=hipSuccess)
        {
            fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
            exit(-1);
        }

        clock_t end_d = clock();

        hipMemcpy(result_image_data_h, result_image_data_d, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

        saveBitmap("image1.bmp", width, height, result_image_data_h,info);

        clock_t start_h = clock();
    //executing CPU code
        meanFilter_h(inputImage, result_image_data_h1, width, height, window_size);
        clock_t end_h = clock();

        time_h += (double)(end_h-start_h)/CLOCKS_PER_SEC;
        time_d += (double)(end_d-start_d)/CLOCKS_PER_SEC;

        hipFree(image_data_d);
        hipFree(result_image_data_d);
    }

    printf("Average GPU execution time: %f\n",(time_d/numOfRounds));
    printf("Average CPU execution time: %f\n",(time_h/numOfRounds));
    printf("CPU/GPU time: %f\n",(time_h/time_d));

    return 0;
}


Output
Begin......
File image1.bmp opened successfully for writing.
Average GPU execution time: 0.001159
Average CPU execution time: 0.041736
CPU/GPU time: 36.010354
														
 								
Original Image



 							
Blurred Image
